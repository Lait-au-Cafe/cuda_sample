#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "kernel.h"
#include <hip/hip_vector_types.h>

__global__ void devAddArrays(
	float *input1,
	float *input2,
	float *result,
	int length
) {
	const int tx = blockIdx.x*blockDim.x + threadIdx.x;

	if(tx >= length){ return; }
	
	result[tx] = input1[tx] + input2[tx];
}

void AddArrays(
	float *d_input1,
	float *d_input2,
	float *d_result,
	int length
){
	// define thread / block size
	dim3 dimBlock(32, 1, 1);
	dim3 dimGrid(
			(length - 1) / dimBlock.x + 1, 
			1, 
			1);

	std::cout
		<< "\n== Configs of " << __func__ << " ==\n"
		<< "Length : " << length << "\n"
		<< "Dim of Grid : (" 
			<< dimGrid.x << ", " 
			<< dimGrid.y << ", " 
			<< dimGrid.z << ")\n"
		<< "Dim of Block : (" 
			<< dimBlock.x << ", " 
			<< dimBlock.y << ", " 
			<< dimBlock.z << ")\n"
		<< std::endl;

	devAddArrays<<<dimGrid, dimBlock, 0 >>>(d_input1, d_input2, d_result, length);
	return;
}
